#include "hip/hip_runtime.h"
#include <iostream>
#include <Windows.h>

#include <reactor/cuda/context.hpp>
#include <reactor/cuda/memory.hpp>
#include <reactor/cuda/stream.hpp>

__global__ void da(int * const a) { *a = 3; }
__global__ void db(int * const b) { *b = 7; }

__global__ void dc(int const * const a, int const * const b, int * const c)
{
	*c = *a + *b;
}

__global__ void dd(int const * const a, int const * const b, int * const d)
{
	*d = *a * *b;
}

int main()
try
{
	std::ios::sync_with_stdio(false);
	{
		reactor::cuda::context _;
		{
#if 0
			reactor::cuda::stream s1;
			reactor::cuda::stream s2;
			reactor::cuda::stream s3;
			reactor::cuda::stream s4;

			reactor::cuda::event e1;
			reactor::cuda::event e2;

			auto d_a = reactor::cuda::malloc<int>(1);
			auto d_b = reactor::cuda::malloc<int>(1);
			auto d_c = reactor::cuda::malloc<int>(1);
			auto d_d = reactor::cuda::malloc<int>(1);

			int a = -1;
			int b = -1;
			int c = -1;
			int d = -1;

			{
				da<<<1, 1, 0, s1.get()>>>(d_a.get());
				s1.notify(e1);
				reactor::cuda::memcpy(&a, d_a.get(), 1, s1.get());
			}
			{
				db<<<1, 1, 0, s2.get()>>>(d_b.get());
				s2.notify(e2);
				reactor::cuda::memcpy(&b, d_b.get(), 1, s2.get());
			}
			{
				s3.wait(e1);
				s3.wait(e2);
				dc<<<1, 1, 0, s3.get()>>>(d_a.get(), d_b.get(), d_c.get());
				reactor::cuda::memcpy(&c, d_c.get(), 1, s3.get());
			}
			{
				s4.wait(e1);
				s4.wait(e2);
				dd<<<1, 1, 0, s4.get()>>>(d_a.get(), d_b.get(), d_d.get());
				reactor::cuda::memcpy(&d, d_d.get(), 1, s4.get());
			}
			{
				s1.synchronize();
				s2.synchronize();
				s3.synchronize();
				std::cout << a << " + " << b << " = " << c << '\n';
			}
			{
				s1.synchronize();
				s2.synchronize();
				s4.synchronize();
				std::cout << a << " * " << b << " = " << d << '\n';
			}
#else
			reactor::cuda::stream sc; // compuate-stream
			reactor::cuda::stream st; // transfer-stream

			reactor::cuda::event ea;
			reactor::cuda::event eb;
			reactor::cuda::event ec;
			reactor::cuda::event ed;

			auto d_a = reactor::cuda::malloc<int>(1);
			auto d_b = reactor::cuda::malloc<int>(1);
			auto d_c = reactor::cuda::malloc<int>(1);
			auto d_d = reactor::cuda::malloc<int>(1);

			int a = -1;
			int b = -1;
			int c = -1;
			int d = -1;

			da<<<1, 1, 0, sc.get()>>>(d_a.get()                      ); sc.notify(ea);
			db<<<1, 1, 0, sc.get()>>>(d_b.get()                      ); sc.notify(eb);
			dc<<<1, 1, 0, sc.get()>>>(d_a.get(), d_b.get(), d_c.get()); sc.notify(ec);
			dd<<<1, 1, 0, sc.get()>>>(d_a.get(), d_b.get(), d_d.get()); sc.notify(ed);

			st.wait(ea); reactor::cuda::memcpy(&a, d_a.get(), 1, st.get());
			st.wait(eb); reactor::cuda::memcpy(&b, d_b.get(), 1, st.get());
			st.wait(ec); reactor::cuda::memcpy(&c, d_c.get(), 1, st.get());
			st.wait(ed); reactor::cuda::memcpy(&d, d_d.get(), 1, st.get());

			st.synchronize();
			std::cout << a << " + " << b << " = " << c << '\n';
			std::cout << a << " * " << b << " = " << d << '\n';
#endif
		}
	}
	return 0;
}
catch (reactor::cuda::exception const & e)
{
	std::cerr << e.what() << '\n';
}
